#include "hip/hip_runtime.h"
#include "gpu_patch.h"

#include <sanitizer_patching.h>

#include "gpu_utils.h"

struct gpu_address_comparator {
    __device__
    bool operator()(MemoryRange &l, MemoryRange &r) {
        return l.start <= r.start;
    }
};

static __device__
SanitizerPatchResult CommonCallback(
    void* userdata,
    uint64_t pc,
    void* ptr,
    uint32_t accessSize,
    uint32_t flags,
    MemoryType type)
{
    auto* pTracker = (MemoryAccessTracker*)userdata;

    uint32_t active_mask = __activemask();
    uint32_t laneid = get_laneid();
    uint32_t first_laneid = __ffs(active_mask) - 1;

    int active_threads = __popc(active_mask);
    if (laneid == first_laneid) {
        atomicAdd((unsigned long long int*)&pTracker->accessCount, (unsigned long long int) active_threads);
    }

    if (pTracker->access_state != nullptr) {
        MemoryAccessState* states = (MemoryAccessState*) pTracker->access_state;
        MemoryRange* start_end = states->start_end;
        MemoryRange range = {(uint64_t) ptr, 0};
        uint32_t pos = map_prev(start_end, range, states->size, gpu_address_comparator());

        if (pos != states->size) {
            if (atomic_load(states->touch + pos) == 0) {
                atomic_store(states->touch + pos, 1);
            }
        }
    }
    __syncwarp(active_mask);

    if (pTracker->tensor_access_state != nullptr) {
        TensorAccessState* tensor_states = (TensorAccessState*) pTracker->tensor_access_state;
        MemoryRange* start_end = tensor_states->start_end;
        MemoryRange range = {(uint64_t) ptr, 0};
        uint32_t tensor_pos = map_prev(start_end, range, tensor_states->size, gpu_address_comparator());

        if (tensor_pos != tensor_states->size) {
            if (atomic_load(tensor_states->touch + tensor_pos) == 0) {
                atomic_store(tensor_states->touch + tensor_pos, 1);
            }
        }
    }
    __syncwarp(active_mask);

    return SANITIZER_PATCH_SUCCESS;
}

extern "C" __device__ __noinline__
SanitizerPatchResult MemoryGlobalAccessCallback(
    void* userdata,
    uint64_t pc,
    void* ptr,
    uint32_t accessSize,
    uint32_t flags,
    const void *pData)
{
    return CommonCallback(userdata, pc, ptr, accessSize, flags, MemoryType::Global);
}

extern "C" __device__ __noinline__
SanitizerPatchResult MemorySharedAccessCallback(
    void* userdata,
    uint64_t pc,
    void* ptr,
    uint32_t accessSize,
    uint32_t flags,
    const void *pData)
{
    return CommonCallback(userdata, pc, ptr, accessSize, flags, MemoryType::Shared);
}

extern "C" __device__ __noinline__
SanitizerPatchResult MemoryLocalAccessCallback(
    void* userdata,
    uint64_t pc,
    void* ptr,
    uint32_t accessSize,
    uint32_t flags,
    const void *pData)
{
    return CommonCallback(userdata, pc, ptr, accessSize, flags, MemoryType::Local);
}

extern "C" __device__ __noinline__
SanitizerPatchResult MemcpyAsyncCallback(void* userdata, uint64_t pc, void* src, uint32_t dst, uint32_t accessSize)
{
    if (src)
    {
        CommonCallback(userdata, pc, src, accessSize, SANITIZER_MEMORY_DEVICE_FLAG_READ, MemoryType::Global);
    }

    return CommonCallback(userdata, pc, (void*)dst, accessSize, SANITIZER_MEMORY_DEVICE_FLAG_WRITE, MemoryType::Shared);
}
